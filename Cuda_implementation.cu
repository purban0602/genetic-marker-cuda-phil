#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <sys/resource.h>


typedef struct{
	char** name;
	char* chrom_c;
	//int* chrom;
	long* pos;
	//long* c_pos;
	char** rest;
}SNP;

typedef struct{
	char* snp_name;
	int* a_id; //length is the number of animals
	char* ab1; 
	char* ab2;
	int* ab;
}Sample;

int NSNPS;
int NSAMPLES;

__device__ void sort_by_bit(SNP* snps, Sample* samples, int bit);

__device__ long scan(long* x);



void read_files(char* map_path, char* snp_path, char** data_string, char** snps_data){
	
	FILE *fd;
	int err;
	int num_lines = -1;
	char** header_array;
	int i;
	
/***********************Allocate string for header info**********/
	printf("Allocating string for header array...\n");

	header_array = (char**) malloc( 10 * sizeof(char*));
	
	for(i = 0; i < 10; i++){
		header_array[i] = (char*)malloc(100); 	
	}
/*****************************************************************/
	
	fd = fopen(snp_path, "r");
	
	
/*******Getting number of SNP and Sample from header****/
	printf("Getting number of SNPs and Samples from header...\n");

	do {
		err = fscanf(fd, "%[^\n]\n", header_array[++num_lines]);
	} while(err != EOF && num_lines < 10);
	
	
	err = sscanf(header_array[5], "Total SNP	%d", &NSNPS);
	err = sscanf(header_array[7], "Total Sample	%d", &NSAMPLES);
/***********************************************************/
	

	
/*************Getting Final Report Data***********************************/
	printf("Getting final report data...\n");

	//char** data_string;
	
	data_string = (char**) malloc(NSNPS * NSAMPLES * sizeof(char*));
	for(i = 0; i < NSNPS*NSAMPLES; i++){
		data_string[i] = (char*)malloc(100); 	
	}
	
	num_lines  = -1;
	do {
		err = fscanf(fd, "%[^\n]\n", data_string[++num_lines]);
	} while(err != EOF && num_lines < NSNPS*NSAMPLES);
	
	fclose(fd);
/**************************************************************************/

	
/************************Getting MapFile Data******************************/
	printf("Getting mapfile data...\n");

	//char** snps_data;
	char* junk = (char*) malloc(50 * sizeof(char));
	
	snps_data = (char**) malloc(NSNPS * sizeof(char*));
	for(i = 0; i < NSNPS; i++){
		snps_data[i] = (char*)malloc(100); 	
	}
	
	fd = fopen(map_path, "r");
	
	int num_lines2 = -1;
	err = fscanf(fd, "%[^\n]\n", junk);
	do {
		err = fscanf(fd, "%[^\n]\n", snps_data[++num_lines2]);
	} while(err != EOF && num_lines2 < NSNPS);
	
	free(junk);
	
	fclose(fd);
/**************************************************************************/
	

}

/*************functions for the radix sort**********************************/

__device__ void radixsort(SNP* snps, Sample* samples){
	
	for(int i = 0; i < 64; i++){
		sort_by_bit(snps, samples, i);
		__syncthreads();
	}
	
}

__device__ void sort_by_bit(SNP* snps, Sample* samples, int bit){
	
		int i = threadIdx.x;
		int size = blockDim.x;
		int index;
		
		/***temperary variables for the snps*****/
		long t_pos = snps->pos[i];
		char* t_name = snps->name[i];
		char t_chrom_c = snps->chrom_c[i];
		//char* t_rest = snps->rest[i];
		Sample t_sample = samples[i];
		
		int p_i = (t_pos >> bit) & 1;
		
		snps->pos[i] = p_i;
		
		__syncthreads();
		
		int ones_before = scan(snps->pos);
		int ones_total = snps->pos[size -1];
		int zeros_total = size - ones_total;
		
		__syncthreads();
		
		if(p_i)
			index = ones_before - 1 + zeros_total;
		else
			index = i - ones_before;
		
		snps->pos[index] = t_pos;
		snps->name[index] = t_name;
		snps->chrom_c[index] = t_chrom_c;
		//snps->rest[index] = t_rest;
		samples[index] = t_sample;
}

/**************************************************************************/

__device__ long scan(long* x){
	
	int i = threadIdx.x;
	int n = blockDim.x;
	int offset;
	
	for ( offset = 1; offset < n; offset *= 2){
		long temp;
		if (i >= offset)
			temp = x[i-offset];
		
		__syncthreads();
		
		if(i >= offset)
			x[i] = temp + x[i];
		
		__syncthreads();
	}
	
	return x[i];
}


void parse(SNP* snps, Sample* animals, char** data_string, char** snp_data){
	
	int i, j, err;
	
	snps->name = (char**) malloc(NSNPS * sizeof(char*));
	snps->chrom_c = (char*) malloc(NSNPS * sizeof(char));
	snps->pos = (long*) malloc(NSNPS * sizeof(long));
	
	for(i = 0; i < NSNPS; i++)
		snps->name[i] = (char*) malloc(50 * sizeof(char));
	
	animals = (Sample*) malloc(NSNPS * sizeof(Sample));
	
	for(i = 0; i < NSNPS; i++){
		animals[i].snp_name = (char*) malloc(50 * sizeof(char));
		animals[i].a_id = (int*) malloc(NSAMPLES * sizeof(int));
		animals[i].ab1 = (char*) malloc(NSAMPLES * sizeof(char));
		animals[i].ab2 = (char*) malloc(NSAMPLES * sizeof(char));
		animals[i].ab = (int*) malloc(NSAMPLES * sizeof(char));
	}
	
	for (i = 0; i < NSNPS; i++){
		err = sscanf(snp_data[i], "%*d	%s	%c	%ld	%*s", 
					  snps->name[i], snps->chrom_c[i], snps->pos[i], snps->rest[i]);
	}
	
	for(i = 0; i < NSNPS; i++){
		for(j = 0; j < NSAMPLES; j++)
			err = sscanf(data_string[i], "%s/t%d/t%*c/t%*c/t%*c/t%*c/t%c/t%c/t%*s", 
							animals[i].snp_name, animals[i].a_id[j], animals[i].ab1[j], animals[i].ab2[j]);
	}
}

__global__ void sort(SNP* snps, Sample* samples, int nsamples){
	
	int id = threadIdx.x;
	radixsort(snps, samples);
	
	for(int i = 0; i < nsamples; i++){
		if (samples[id].ab1[i] == 'A' && samples[id].ab2[i] == 'A'){
			samples[id].ab[i] = 1;
		}else if(samples[id].ab1[i] == 'B' && samples[id].ab2[i] == 'B'){
			samples[id].ab[i] = 2;
		}else{
			samples[id].ab[i] = 3;
		}
	}
}
int main(int argc, char** argv){
	printf("Begin.\n");
	
	SNP h_snps;
/*	
	typedef struct{
		char** name;
		char* chrom_c;
		//int* chrom;
		long* pos;
		//long* c_pos;
		//char** rest;
	}SNP;
*/
	Sample* h_samples;
/*
	typedef struct{
		char* snp_name;
		int* a_id; //length is the number of animals
		char* ab1; 
		char* ab2;
		int* ab;
	}Sample;
*/
	//char map_path[], snp_path[];
	char** data_string, **snps_data;
	char** d_name;
	char* d_chrom_c;
	long* d_pos;

	printf("Reading files...\n");
	
	//map_path = argv[1];
	char map_path[] = "./sample-files/test-files/SNP_Map_Truncated.txt";	
	//snp_path = argv[2];
	char snp_path[] = "./sample-files/test-files/FinalReport_Truncated.txt";
	
	read_files(map_path, snp_path, data_string, snps_data);

	printf("Files read.\nParsing...\n");

	parse(&h_snps, h_samples, data_string, snps_data);

	printf("Data parsed.\n");
	
	free(data_string);
	free(snps_data);
	
	printf("Allocating CUDA memory...\n");

	hipMalloc((void**)&(d_pos), sizeof(long)*NSNPS);
	hipMalloc((void**)&(d_chrom_c), sizeof(char)*NSNPS);
	hipMalloc((void**)d_name, sizeof(char*)*NSNPS);
	
	hipMemcpy(d_pos, (h_snps.pos), sizeof(long)*NSNPS, hipMemcpyHostToDevice);
	hipMemcpy(d_chrom_c, (h_snps.chrom_c), sizeof(char)*NSNPS, hipMemcpyHostToDevice);
	hipMemcpy(d_chrom_c, (h_snps.chrom_c), sizeof(char)*NSNPS, hipMemcpyHostToDevice);
	
}
